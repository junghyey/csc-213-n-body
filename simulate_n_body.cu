

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>

#define G 10

// (6.67430 * (pow (10, -11)))
/**
 * Simulate n body problem
 *  
 */


typedef struct body { 
    double mass;
    double position[3];
    double velocity[3];
    double net_force[3];
} body_t;

__host__ __device__ body_t *n_bodies;

__host__ __device__ double (*forces) [3];


/**
 * Function to read input
 */



/**
 * Force calculation
 * force applied to index1 due to index2
 * (1,2)
 * column : body that exerting force
 * row : body that is "affected"by the force of the column
 */

__global__ void calculate_force ( int N){
    size_t index1 = threadIdx.x;
    size_t index2 = threadIdx.y;
    if (index1 == index2)
    {
        forces[index1 * N + index2][0] = 0;
        forces[index1 * N + index2][1] = 0;
        forces[index1 * N + index2][2] = 0;
        return;
    }
    // calculate magnitude
    body_t body1 = n_bodies[index1];
    body_t body2 = n_bodies[index2];
    
    double dx = body2.position[0] - body1.position[0];
    double dy = body2.position[1] - body1.position[1];
    double dz = body2.position[2] - body1.position[2];

    double distance = sqrt(pow(dx, 2) + pow(dy, 2) + pow(dz, 2));

    double F_mag = (G * body1.mass * body2.mass)/(pow(distance,2));

    double F_x = F_mag * (dx / distance);
    double F_y = F_mag * (dy / distance);
    double F_z = F_mag * (dz / distance);

    printf("body %d\n", index1);
    printf("F_x: %lf, F_y: %lf, F_z: %lf\n", F_x, F_y, F_z);

    forces[index1 * N + index2][0] = F_x;
    forces[index1 * N + index2][1] = F_y;
    forces[index1 * N + index2][2] = F_z;
}

__global__ void net_force (int N){

    size_t index = threadIdx.x;

    int start_index = index * N;
    double F_x =0.0;
    double F_y = 0.0;
    double F_z = 0.0;
    for (int i = start_index; i < start_index + N ; i++)
    {
        F_x += forces[i][0];
        F_y += forces[i][1];
        F_z += forces[i][2];
    }
    printf("net force body %d\n", index);
    printf("F_x: %lf, F_y: %lf, F_z: %lf\n", F_x, F_y, F_z);
    n_bodies[index].net_force[0] = F_x;
    n_bodies[index].net_force[1] = F_y;
    n_bodies[index].net_force[2] = F_z;

}
/**
 * Update funciton (euler's method)
 */

__global__  void update_body ( double time_step){
     size_t index = threadIdx.x;
    body_t body = n_bodies[index];
    double mass = body.mass;
    double f_X = body.net_force[0];
    double f_Y = body.net_force[1];
    double f_Z = body.net_force[2];

    double v_X = body.velocity[0];
    double v_Y = body.velocity[1];
    double v_Z = body.velocity[2];
  
    double a_X = f_X / mass;
    double a_Y = f_Y / mass;
    double a_Z = f_Z / mass;

// velocity
    n_bodies[index].velocity[0] = v_X + a_X * time_step;
    n_bodies[index].velocity[1] = v_Y + a_Y * time_step;
    n_bodies[index].velocity[2] = v_Z + a_Z * time_step;

// position

    n_bodies[index].position[0] = body.position[0] +  n_bodies[index].velocity[0] * time_step;
    n_bodies[index].position[1] = body.position[1] +  n_bodies[index].velocity[1] * time_step;
    n_bodies[index].position[2] = body.position[2] +  n_bodies[index].velocity[2] * time_step;

}

void print_object(int index){

    printf("Body %d\n", index);
    printf("Position:\n");
    for (int i = 0; i < 3; i++)
    {
        printf("%lf, ", n_bodies[index].position[i]);
    }
    printf("\n");
    printf("Velocity:\n");
    for (int i = 0; i < 3; i++)
    {
        printf("%lf, ", n_bodies[index].velocity[i]);
    }
    printf("\n");
    printf("Net Force:\n");
    for (int i = 0; i < 3; i++)
    {
        printf("%lf, ", n_bodies[index].net_force[i]);
    }
    printf("\n");

}

void simulate_n_body (double time_step, int N,  double (*forces) [3], body_t *n_bodies){

//allocate memory & copy to gpu
double(*gpu_force)[3];
    if (hipMalloc(&gpu_force, sizeof(N * N* sizeof(double[3]))) != hipSuccess){
        fprintf(stderr, "Failed to allocate memory for the force");
    }

    body_t *n_bodies_gpu;
    if (hipMalloc(&n_bodies_gpu, sizeof(sizeof(body_t) * N)) != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate memory for the bodies");
    }

    if(hipMemcpy(gpu_force, forces, sizeof(N * N* sizeof(double[3])), hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "Failed to copy force to the GPU\n");
  }

   if(hipMemcpy(n_bodies_gpu, n_bodies, sizeof(sizeof(body_t) * N), hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "Failed to copy n bodies to the GPU\n");
  } 


  // calculate
  calculate_force<<<N, N>>>(N);
  if (hipDeviceSynchronize() != hipSuccess)
  {
      fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(hipPeekAtLastError()));
  }

//     net_force<<<1,N>>>(N);
// //      if(cudaDeviceSynchronize() != cudaSuccess) {
// //     fprintf(stderr, "CUDA Error: %s\n", cudaGetErrorString(cudaPeekAtLastError()));
// //   }
//     update_body<<<1, N>>>(time_step);
//      if(cudaDeviceSynchronize() != cudaSuccess) {
//     fprintf(stderr, "CUDA Error: %s\n", cudaGetErrorString(cudaPeekAtLastError()));
//   }



//copy back to CPU

    if(hipMemcpy(forces, gpu_force, sizeof(N * N* sizeof(double[3])), hipMemcpyDeviceToHost) != hipSuccess) {
        fprintf(stderr, "Failed to copy forces from the GPU\n");
  }

 if(hipMemcpy(n_bodies, n_bodies_gpu, sizeof(sizeof(body_t) * N), hipMemcpyDeviceToHost) != hipSuccess) {
    fprintf(stderr, "Failed to copy n bodies from the GPU\n");
  }

  hipFree(gpu_force);
hipFree(n_bodies_gpu);


}




int main(int argc, char** argv){
    int N = 2;
    n_bodies = (body_t *)malloc(sizeof(body_t) * N);
    forces = (double(*)[3])malloc(N * N* sizeof(double[3]));
    body_t body1 = {.mass = 500, .position = {0, 0, 0}, .velocity = { 0, 0, 0 }};
    body_t body2 = {.mass = 100, .position = {4, 3, 0}, .velocity = { 0, 0, 0 }};

    n_bodies[0] = body1;
    n_bodies[1] = body2;

    double time_step = 0.01;

    simulate_n_body(time_step, N, forces, n_bodies);

    // calculate_force<<<N, N>>>(N);
    // calculate_force(0, 0, N);
    // calculate_force(1, 1, N);

    // calculate_force(0, 1, N);
    // calculate_force(1, 0, N);
   

    for (int i = 0; i < N * N; i++){
        printf("%d\n", i);
        for (int j = 0; j < 3; j++)
        {

            printf("%lf", i, forces[i][j]);
        }
        printf("\n");
    }

  

   
    // net_force(0, N);
    // net_force(1, N);
   // printf("==========afer net force ===================\n");
  //  print_object(0);
    //print_object(1);

    //printf("==========afer update ===================\n");

   // update_body<<<1, N>>>(time_step);
    //print_object(0);

    return 0;
}